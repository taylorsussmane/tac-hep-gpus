
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>

#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

	// Use atomicAdd
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < N){
		atomicAdd(C, A[idx] * B[idx]);
	}	
}


int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;

	// Allocate device memory
	hipMalloc(&d_A, DSIZE*sizeof(int));
	hipMalloc(&d_B, DSIZE*sizeof(int));
	hipMalloc(&d_C, sizeof(int));	
	
	// Check memory allocation for errors
	cudaCheckErrors();

	// Copy the matrices on GPU
	hipMemcpy(d_A, h_A, DSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, DSIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeof(int), hipMemcpyHostToDevice);

	// Check memory copy for errors
	cudaCheckErrors();

	// Define block/grid dimentions and launch kernel
	int gridSize = DSIZE/BLOCK_SIZE;
	dot_product<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C, DSIZE);

	// Copy results back to host
        hipMemcpy(h_C, d_C, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	// Check copy for errors
	cudaCheckErrors();

	// Verify result
	std::cout<< "Dot product: "<<std::endl;
	std::cout<<"A vector is a vector of size "<<DSIZE<< " where all components have value " << a<<std::endl;
	std::cout<< "B vector is a vector of size "<<DSIZE << " where all components have value "<< b<< std::endl;
	std::cout<< "A \u22C5 B = "<< *h_C << std::endl;

	// Free allocated memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;

}
