
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>

using namespace std;

#define N 64
#define RADIUS 2
#define BLOCK_SIZE 32


__global__ void stencil_2d(int *in, int *out) {

	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS][BLOCK_SIZE + 2 * RADIUS];
	int gindex_x = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex_x = threadIdx.x + RADIUS;
	int gindex_y = threadIdx.y + blockIdx.y * blockDim.y;
	int lindex_y = threadIdx.y + RADIUS;

	// Read input elements into shared memory
	int size = N + 2 * RADIUS;
	temp[lindex_x][lindex_y] = in[gindex_y+size*gindex_x];

	if (threadIdx.x < RADIUS) {
		temp[lindex_x-RADIUS][lindex_y] = in[size*(gindex_x - RADIUS)+gindex_y];
		temp[lindex_x+BLOCK_SIZE][lindex_y] = in[size*(gindex_x+BLOCK_SIZE)+gindex_y];
	}

	if (threadIdx.y < RADIUS ) {
		temp[lindex_x][lindex_y-RADIUS] = in[gindex_x*size+(gindex_y-RADIUS)];
		temp[lindex_x][lindex_y+BLOCK_SIZE] = in[gindex_x*size+(gindex_y+BLOCK_SIZE)];
	}

	__syncthreads();

	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS; offset <= RADIUS; offset++){
		result += temp[lindex_x+offset][lindex_y+offset];
	}

	__syncthreads();
	// Store the result
	out[gindex_y+size*gindex_x] = result;
}


void fill_ints(int *x, int n) {
   // Store the result
   // https://en.cppreference.com/w/cpp/algorithm/fill_n
   fill_n(x, n, 1);
}


int main(void) {

	int *in, *out; // host copies of a, b, c
	int *d_in, *d_out; // device copies of a, b, c

	// Alloc space for host copies and setup values
	int size = (N + 2*RADIUS)*(N + 2*RADIUS) * sizeof(int);
	in = (int *)malloc(size); fill_ints(in, (N + 2*RADIUS)*(N + 2*RADIUS));
	out = (int *)malloc(size); fill_ints(out, (N + 2*RADIUS)*(N + 2*RADIUS));

	// Alloc space for device copies
	hipMalloc((void **)&d_in, size);
	hipMalloc((void **)&d_out, size);

	// Copy to device
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

	// Launch stencil_2d() kernel on GPU
	int gridSize = (N + BLOCK_SIZE-1)/BLOCK_SIZE;
	dim3 grid(gridSize, gridSize);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	// Launch the kernel 
	// Properly set memory address for first element on which the stencil will be applied
	stencil_2d<<<grid,block>>>(d_in + RADIUS*(N + 2*RADIUS) + RADIUS , d_out + RADIUS*(N + 2*RADIUS) + RADIUS);

	// Copy result back to host
	hipMemcpy(in, d_in, size, hipMemcpyDeviceToHost);
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

	// Error Checking
	for (int i = 0; i < N + 2 * RADIUS; ++i) {
		for (int j = 0; j < N + 2 * RADIUS; ++j) {

			if (i < RADIUS || i >= N + RADIUS) {
				if (out[j+i*(N + 2 * RADIUS)] != 1) {
					printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], 1);
					return -1;
				}
			}
			else if (j < RADIUS || j >= N + RADIUS) {
				if (out[j+i*(N + 2 * RADIUS)] != 1) {
					printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], 1);
					return -1;
				}
			}		 
			else {
				if (out[j+i*(N + 2 * RADIUS)] != 1 + 4 * RADIUS) {
					printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], 1 + 4*RADIUS);
					return -1;
				}
			}
		}
	}

	// Cleanup
	free(in);
	free(out);
	hipFree(d_in);
	hipFree(d_out);
	printf("Success!\n");

	return 0;
}


