#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <iostream>

using namespace std;

#define N 64
#define RADIUS 2
#define BLOCK_SIZE 32


__global__ void stencil_2d(int *in, int *out) {

	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS][BLOCK_SIZE + 2 * RADIUS];
	int gindex_x = FIXME
	int lindex_x = FIXME
	int gindex_y = FIXME
	int lindex_y = FIXME

	// Read input elements into shared memory
	int size = N + 2 * RADIUS;
	temp[lindex_x][lindex_y] = FIXME

	if (threadIdx.x < RADIUS) {
		FIXME
	}

	if (threadIdx.y < RADIUS ) {
		FIXME
	}


	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS; offset <= RADIUS; offset++){
		FIXME
	}

	FIXME
	// Store the result
	out[gindex_y+size*gindex_x] = result;
}


void fill_ints(int *x, int n) {
   // Store the result
   // https://en.cppreference.com/w/cpp/algorithm/fill_n
   fill_n(x, n, 1);
}


int main(void) {

	int *in, *out; // host copies of a, b, c
	int *d_in, *d_out; // device copies of a, b, c

	// Alloc space for host copies and setup values
	int size = (N + 2*RADIUS)*(N + 2*RADIUS) * sizeof(int);
	in = (int *)malloc(size); fill_ints(in, (N + 2*RADIUS)*(N + 2*RADIUS));
	out = (int *)malloc(size); fill_ints(out, (N + 2*RADIUS)*(N + 2*RADIUS));

	// Alloc space for device copies
	hipMalloc((void **)&d_in, size);
	FIXME

	// Copy to device
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	FIXME

	// Launch stencil_2d() kernel on GPU
	int gridSize = (N + BLOCK_SIZE-1)/BLOCK_SIZE;
	dim3 grid(gridSize, gridSize);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	// Launch the kernel 
	// Properly set memory address for first element on which the stencil will be applied
	stencil_2d<<<grid,block>>>(d_in + RADIUS*(N + 2*RADIUS) + RADIUS , d_out + RADIUS*(N + 2*RADIUS) + RADIUS);

	// Copy result back to host
	FIXME

	// Error Checking
	for (int i = 0; i < N + 2 * RADIUS; ++i) {
		for (int j = 0; j < N + 2 * RADIUS; ++j) {

			if (i < RADIUS || i >= N + RADIUS) {
				if (out[j+i*(N + 2 * RADIUS)] != 1) {
					printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], 1);
					return -1;
				}
			}
			else if (j < RADIUS || j >= N + RADIUS) {
				if (out[j+i*(N + 2 * RADIUS)] != 1) {
					printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], 1);
					return -1;
				}
			}		 
			else {
				if (out[j+i*(N + 2 * RADIUS)] != 1 + 4 * RADIUS) {
					printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], 1 + 4*RADIUS);
					return -1;
				}
			}
		}
	}

	// Cleanup
	free(in);
	free(out);
	hipFree(d_in);
	hipFree(d_out);
	printf("Success!\n");

	return 0;
}


