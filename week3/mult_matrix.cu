
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const int DSIZE = 256;
const float A_val = 3.0f;
const float B_val = 2.0f;

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)

// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const float *A, const float *B, float *C, int size) {
	for (int i = 0; i<size; i++){
		for (int j = 0; j<size; j++){
			float tempSum = 0;

			for (int k = 0; k<size; k++){
				tempSum += A[i*size+k]+B[k*size+j];
			}

			C[i*size+j] = tempSum;
		}
	} 
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {

    // create thread x index
    // create thread y index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        float tempSum = 0;
        for (int i = 0; i < size; i++){
	    tempSum += A[idy*size + i]*B[i*size + idx];
        }
        C[idy*size+idx] = tempSum;                    
    }
}

int main() {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // These are used for timing
    clock_t t0, t1, t2, t3;
    double t1sum=0.0;
    double t2sum=0.0;
    double t3sum=0.0;

    // start timing
    t0 = clock();

    // N*N matrices defined in 1 dimention
    // If you prefer to do this in 2-dimentions cupdate accordingly
    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data from host to device
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
	hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
	hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));

	hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  

    // Launch kernel
    // Specify the block and grid dimentions 
    dim3 block(DSIZE, DSIZE);
    dim3 grid(1, 1);
    matrix_mul_gpu<<<grid, block>>>(d_A, d_B, d_C, DSIZE);

    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. GPU compute took %f seconds\n", t2sum);

    // Excecute and time the cpu matrix multiplication function
    matrix_mul_cpu(h_A, h_B, h_C, DSIZE);

    // CPU timing
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf ("Done. CPU compute took %f seconds\n", t3sum);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C); 
    
    return 0;

}
